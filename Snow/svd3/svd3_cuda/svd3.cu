#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "svd3_cuda.h"
#include <stdio.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
  # error printf is only supported on devices of compute capability 2.0 and higher, please compile with -arch=sm_20 or higher
#endif

extern "C" void runCudaPart();

__host__ __device__ __forceinline__
void printMat3(float a11, float a12, float a13,
               float a21, float a22, float a23,
               float a31, float a32, float a33)
{
    printf("%f %f %f \n", a11, a12, a13);
    printf("%f %f %f \n", a21, a22, a23);
    printf("%f %f %f \n", a31, a32, a33);
}

__global__ void svd3_test()
{
    int tid = blockIdx.x;

    float a11, a12, a13, a21, a22, a23, a31, a32, a33;

    a11= -0.558253; a12 = -0.0461681; a13 = -0.505735;
    a21 = -0.411397; a22 = 0.0365854; a23 = 0.199707;
    a31 = 0.285389; a32 =-0.313789; a33 = 0.200189;

    float u11, u12, u13, u21, u22, u23, u31, u32, u33;
    float s11, s12, s13, s21, s22, s23, s31, s32, s33;
    float v11, v12, v13, v21, v22, v23, v31, v32, v33;

    svd(a11, a12, a13, a21, a22, a23, a31, a32, a33,
    u11, u12, u13, u21, u22, u23, u31, u32, u33,
    s11, s12, s13, s21, s22, s23, s31, s32, s33,
    v11, v12, v13, v21, v22, v23, v31, v32, v33);
}

void runCudaPart()
{
    // all your cuda code here
    svd3_test<<<1,1>>>(); // 5 blocks, 1 GPU thread each
    hipDeviceSynchronize();
}
